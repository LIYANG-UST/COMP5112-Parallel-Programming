#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>

#include <cmath>
using namespace std;

const int TILE_WIDTH = 16;
__global__ void MatrixMulKernel(int *d_M,int *d_N,int *d_P,int m,int n,int k)
{
  __shared__ int ds_M[TILE_WIDTH][TILE_WIDTH];
  __shared__ int ds_N[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  //Identify the row and column of the Pd element to work on
  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;

  int pValue = 0;

  //loop over the Md and Nd tiles required to comput the Pd element
  for(int t = 0; t < (n-1) / TILE_WIDTH + 1; ++t)
  {
	if(row < m && t * TILE_WIDTH + tx < n)
	  ds_M[ty][tx] = d_M[row * n + t * TILE_WIDTH + tx];
	else
	  ds_M[ty][tx] = 0;

	if(col < k && t * TILE_WIDTH + ty < n)
	  ds_N[ty][tx] = d_N[(t * TILE_WIDTH + ty) * k + col];
	else
	  ds_N[ty][tx] = 0;
	__syncthreads();

	for(int i = 0; i < TILE_WIDTH; ++i)
	  pValue += ds_M[ty][i] * ds_N[i][tx];
	__syncthreads();
  }
  if(row < m && col < k)
	d_P[row * k + col] = pValue;
}

int main()
{
  //freopen("out","w",stdout);
  int m = 600, n = 700, k = 1000;
  int *h_M, *h_N, *d_M, *d_N;
  int *h_P, *d_P;
  size_t sizeM = m * n * sizeof(int);
  size_t sizeN = n * k * sizeof(int);
  size_t sizeP = m * k * sizeof(int);

  h_M = (int *) malloc(sizeM);
  h_N = (int *) malloc(sizeN);
  h_P = (int *) malloc(sizeP);

  hipMalloc(&d_M,sizeM);
  hipMalloc(&d_N,sizeN);
  hipMalloc(&d_P,sizeP);

  for(int i = 0; i < m * n; ++i)
  {
	if(i % 2 == 0)
	  h_M[i] = 1;
	else
	  h_M[i] = 0;
  }

  for(int i = 0;i < n * k; ++i)
  {
	if(i % 2 == 0)
	  h_N[i] = 0;
	else
	  h_N[i] = 1;
  }

  hipMemcpy(d_M,h_M,sizeM,hipMemcpyHostToDevice);
  hipMemcpy(d_N,h_N,sizeN,hipMemcpyHostToDevice);

  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);

  dim3 grid((int)ceil(k*1.0 / TILE_WIDTH), (int)ceil(m*1.0/ TILE_WIDTH));
  dim3 block(TILE_WIDTH,TILE_WIDTH);
  MatrixMulKernel<<<grid,block>>>(d_M,d_N,d_P,m,n,k);

  hipEventRecord(stop,0);
  //cudaDeviceSynchronize();
  hipEventSynchronize(stop);
  float ElapsedTime;
  hipEventElapsedTime(&ElapsedTime,start,stop);
  printf("Kernel Elpased Time: %.3f ms\n",ElapsedTime);

  hipMemcpy(h_P,d_P,sizeP,hipMemcpyDeviceToHost);
/*  
for(int i = 0; i < m * k; ++i)
	printf("%d\n",h_P[i]);
  printf("\n");
*/

 free(h_P);
 free(h_M);
 free(h_N);
 hipFree(d_P);
 hipFree(d_M);
 hipFree(d_N);

  return 0;
}
